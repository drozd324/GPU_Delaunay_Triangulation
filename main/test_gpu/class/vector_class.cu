
#include <hip/hip_runtime.h>
#include <stdio.h>

int N = 10;

__global__ void add_vectors(double* a, double* b, int n, double* c) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id < n) { 
		c[id] = a[id] + b[id];
	}
}


void print_array(double* array, int n) {
	for (int i=0; i<n; ++i) {
		printf("%lf ", array[i]);
	}
	printf("\n");
}


struct VectorAdder {
    double *a_d, *b_d, *c_d;
    int size;

    VectorAdder(double* a, double* b, int n) :  size(n) {
		hipMalloc(&a_d, size * sizeof(double));
		hipMalloc(&b_d, size * sizeof(double));
		hipMalloc(&c_d, size * sizeof(double));

		hipMemcpy(a_d, a, size * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(b_d, b, size * sizeof(double), hipMemcpyHostToDevice);

		compute();
    }

    ~VectorAdder() {
		hipFree(a_d);
		hipFree(b_d);
		hipFree(c_d);
    }


    void compute() {
		// Set execution configuration parameters
		//      thr_per_blk: number of CUDA threads per grid block
		//      blk_in_grid: number of blocks in grid
		int thr_per_blk = 256;
		int blk_in_grid = ceil( float(size) / thr_per_blk );
		add_vectors<<<blk_in_grid, thr_per_blk>>>(a_d, b_d, size, c_d);
    }

    void getResult(double* c) {
		hipMemcpy(c, c_d, (size-2) * sizeof(double), hipMemcpyDeviceToHost);
    }
};


// Main program
int main() {
    // Number of bytes to allocate for N doubles

//    double *a = (double*)malloc(N * sizeof(double));
//    double *b = (double*)malloc(N * sizeof(double));
//    double *c = (double*)malloc(N * sizeof(double));

    double *a = new double[N];
    double *b = new double[N];
    double *c = new double[N];

    // Fill host arrays A and B
    for(int i=0; i<N; i++) {
        a[i] = 1.0;
        b[i] = 2.0;
    }

	VectorAdder vecadd(a, b, N);
	vecadd.getResult(c);

	print_array(a, N);
    print_array(b, N);
	print_array(c, N);

//    free(a);
//    free(b);
//    free(c);

    delete[] a;
    delete[] b;
    delete[] c;
		
    printf("\n---------------------------\n");
    printf("__SUCCESS__\n");
    printf("---------------------------\n");
    printf("N                 = %d\n", N);
//    printf("Threads Per Block = %d\n", thr_per_blk);
//    printf("Blocks In Grid    = %d\n", blk_in_grid);
    printf("---------------------------\n\n");

    return 0;
}
