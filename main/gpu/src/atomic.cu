#include "hip/hip_runtime.h"
#include "atomic.h"

__device__ float atomicAddFloat(float* address, float val) {
	int* address_as_ull = (int*)address;
	int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __float_as_int(val + __int_as_float(assumed)));
	} while (assumed != old);
	return __int_as_float(old);
}

__device__ float atomicMaxFloat(float* address, float val) {
	int* address_as_ull = (int*)address;
	int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __float_as_int(max(val, __int_as_float(assumed))));
	} while (assumed != old);
	return __int_as_float(old);
}

__device__ float atomicMinFloat(float* address, float val) {
	int* address_as_ull = (int*)address;
	int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __float_as_int(min(val, __int_as_float(assumed))));
	} while (assumed != old);
	return __int_as_float(old);
}

