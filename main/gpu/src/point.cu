#include "hip/hip_runtime.h"
#include "point.h"

__host__ __device__ real dist(Point a, Point b) {
	return sqrt( SQR(a.x[0] - b.x[0]) + SQR(a.x[1] - b.x[1]) );
}
