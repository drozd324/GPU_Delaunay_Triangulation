#include "hip/hip_runtime.h"
#include "delaunay.h"


/*
 * Constructor which creates the delaunay triagulation from an array of 'points'
 * and its lenght 'n'.
 */
Delaunay::Delaunay(Point* points, int n) : 
	saveFile("./data/data.txt", std::ios_base::app)
{
	// =============== allocation on host
	npts[0] = n;
	pts = (Point*) malloc((npts[0] + 3) * sizeof(Point));
	ptToTri = (int*) malloc(npts[0] * sizeof(int));

	nTri[0] = 0;
	nTriMax[0] = 2*(npts[0]+3) - 2 - 3;
	triList = (Tri*) malloc(nTriMax[0] * sizeof(Tri));

	for (int i=0; i<npts[0]; i++) {
		pts[i] = points[i];
	}

	// =============== alloc on device
	// alloc points
	hipMalloc(&npts_d,               sizeof(int)  );
	hipMalloc(&pts_d , (npts[0]+3) * sizeof(Point));
	hipMalloc(&ptToTri_d, npts[0] * sizeof(Point));

	// alloc triangles
	hipMalloc(&nTri_d   , sizeof(int));
	hipMalloc(&nTriMax_d, sizeof(int));
	hipMalloc(&triList_d, nTriMax[0] * sizeof(Tri));
	
	// counters
	hipMalloc(&num_tris_to_insert_d, sizeof(int));

	// copying exitsting info to gpu
	hipMemcpy(pts_d    , pts    , npts[0] * sizeof(Point), hipMemcpyHostToDevice);
	hipMemcpy(nTriMax_d, nTriMax,           sizeof(int)  , hipMemcpyHostToDevice);

	initSuperTri();

	// save points data to file
	saveFile << npts[0]+3 << "\n";
	for (int i=0; i<npts[0]+3; ++i) {
		saveFile << pts[i].x[0] << " " << pts[i].x[1] << "\n";
	}
	saveFile << "\n"; 

	saveToFile();
	hipMemcpy(triList, triList_d, nTriMax[0] * sizeof(Tri), hipMemcpyDeviceToHost);
	saveToFile();

	for (int i=0; i<npts; ++i) { 
		std::cout << "============[PASS " << i << "]============ \n"; 

		dim3 threadsPerBlock(32);
		dim3 numBlocks1((*npts)/threadsPerBlock.x + (!((*npts) % threadsPerBlock.x) ? 0:1));

		checkInsert<<<numBlocks1, threadsPerBlock>>>();
		
		// ==== MARK TRIANGLES FOR INSERTION ====
//		int num_to_insert  = checkInsert();
//		std::cout << "num to insert: " << num_to_insert << "\n";

		//hipMemcpy(triList_d, triList, sizeof(Tri), hipMemcpyDeviceToHost);
//
//		// ==== INSERT ====
//		int num_inserted_tri = insert();
//		std::cout << "number of inserted intrangles: " << num_inserted_tri << "\n";
//
//		if (num_inserted_tri == 0) {
//			break; 
//		}

		std::cout << "nTri " << nTri << "/" << nTriMax << "\n";
	}
	//

	// copy everything back to host
		
	hipFree(pts_d);
	hipFree(npts_d);
	hipFree(ptToTri_d);

	hipFree(triList_d);
	hipFree(nTri_d);
	hipFree(nTriMax_d);
	
	free(triList); 
	free(pts);
	free(ptToTri);

//	int nflips = -1;
//	while (nflips != 0) {
//		nflips = legalize();
//		std::cout << "Performed	" << nflips  << " additional flips\n"; 
//	}
//
//	std::cout << "Triangluation is Delaunay\n";

	//saveToFile(true);
}

/*
 * Pick a triangle by index 'i' in triList and insert its center point.
 * Returns the number of a new triangles created. 
 */
//__device__ int Delaunay::insertInTri(int i) {
//	int r = triList[i].center;
//
//	if (r == -1) { // if no points inside this triangle, continue
//		return 0;
//	}
//
//	insertPtInTri(r, i);
//
//	return 2;
//}

/*
 * Inserts a point into triangle indexed by 'i' (splits the triangle into 3 creating 
 * two new triangles) if possible. Returns the number of a new triangles created.
 *
 * @param i Index of triangle in the array triList.
 */
//__device__ int Delaunay::insertPtInTri(int r, int i) {
//
//	int p[3] = {triList[i].p[0],
//				triList[i].p[1],
//				triList[i].p[2]};
//
//	int n[3] = {triList[i].n[0],
//				triList[i].n[1],
//				triList[i].n[2]};
//
//	int o[3] = {triList[i].o[0],
//				triList[i].o[1],
//				triList[i].o[2]};
//
//	int p0[3] = {r, p[0], p[1]};
//	int n0[3] = {nTri+1, n[0], nTri};
//	int o0[3] = {1, o[0], 2};
//
//	int p1[3] = {r, p[1], p[2]};
//	int n1[3] = {i, n[1], nTri+1};
//	int o1[3] = {1, o[1], 2};
//
//	int p2[3] = {r, p[2], p[0]};
//	int n2[3] = {nTri, n[2], i};
//	int o2[3] = {1, o[2], 2};
//
//
//	int nspts = triList[i].nlpts;
//	int* spts = new int[nspts];
//	for (int k=0; k<nspts; ++k) {
//		spts[k] = triList[i].lpts[k];
//	}
//
//	triList[nTri  ].writeTri(pts, npts, spts, nspts, p1, n1, o1);
//	triList[nTri+1].writeTri(pts, npts, spts, nspts, p2, n2, o2);
//	triList[i     ].writeTri(pts, npts, spts, nspts, p0, n0, o0);
//
//	// marking edge for flipping
//	triList[nTri  ].flip = 1;
//	triList[nTri+1].flip = 1;
//	triList[i     ].flip = 1;
//
//	delete[] spts;
//
//	// updates neighbour points opposite point if they exist
//	if (n[0] >= 0) {
//		triList[n[0]].o[(o[0]+1) % 3] = 0;
//		triList[n[0]].n[(o[0]+1) % 3] = i;
//	}
//
//	if (n[1] >= 0) {
//		triList[n[1]].o[(o[1]+1) % 3] = 0;
//		triList[n[1]].n[(o[1]+1) % 3] = nTri;
//	}
//
//	if (n[2] >= 0) {
//		triList[n[2]].o[(o[2]+1) % 3] = 0;
//		triList[n[2]].n[(o[2]+1) % 3] = nTri+1;
//	}
//	
//	nTri += 2;		
//
//	// try to make some ascii art diagrams maybe good for explenation
//	//saveToFile();
//
//	return 2;
//}

/*
 * Inserts a point into triangles which contain points inside of them. The point is 
 * chosen to be the closest to the circumcenter of this triangle if available. This 
 * function also reutrns the number of triangles added into the triangulation.
 */
//__device__ int Delaunay::insert() {
//	int num_inserted_tri = 0;
//
//	int max = nTri;
//	for (int i=0; i<max; ++i) {
//		num_inserted_tri += insertInTri(i);
//	}
//
//	return num_inserted_tri;
//}

/*
 *
 */

void Delaunay::initSuperTri() {

	Point avgPoint[1];
	Point* avgPoint_d;

	// computing the average point
	avgPoint->x[0] = 0;
	avgPoint->x[1] = 0;
	hipMalloc(&avgPoint_d, sizeof(Point));
	hipMemcpy(avgPoint_d, avgPoint, sizeof(Point), hipMemcpyHostToDevice);

	printf("avgPoint before: (%f, %f)\n", avgPoint->x[0], avgPoint->x[1]);

	//dim3 threadsPerBlock(warpSize);
	dim3 threadsPerBlock(32);
	dim3 numBlocks1((2*npts[0])/threadsPerBlock.x + (!((2*npts[0]) % threadsPerBlock.x) ? 0:1));

	sumPoints<<<numBlocks1, threadsPerBlock>>>(pts_d, npts[0], avgPoint_d);

	hipMemcpy(avgPoint, avgPoint_d, sizeof(Point), hipMemcpyDeviceToHost);

	avgPoint->x[0] /= npts[0];
	avgPoint->x[1] /= npts[0];

	printf("avgPoint After: (%f, %f)\n", avgPoint->x[0], avgPoint->x[1]);

	// computing the largest distance bewtween two points
	largest_dist[0] = 0;
	hipMalloc(&largest_dist_d, sizeof(float));
	hipMemcpy(largest_dist_d, largest_dist, sizeof(float), hipMemcpyHostToDevice);
	//hipMemset(largest_dist_d, 0, sizeof(float));

	printf("before largest_dist: %f\n", *largest_dist);

	int ncomps = (npts[0]*(npts[0]-1)) / 2; //number of comparisons
	dim3 numBlocks2(ncomps/threadsPerBlock.x + (!(ncomps % threadsPerBlock.x) ? 0:1));
	computeMaxDistPts<<<numBlocks2, threadsPerBlock>>>(pts_d, npts[0], largest_dist_d);

	hipMemcpy(largest_dist, largest_dist_d, sizeof(float), hipMemcpyDeviceToHost);

	printf("after largest_dist: %f\n", *largest_dist);

	// writing supertriangle points to pts
	float center_x = avgPoint->x[0];
	float center_y = avgPoint->x[1];
	float radius = *largest_dist;

	pts[npts[0]    ] = Point(center_x + radius*1.73205, center_y - radius  );
	pts[npts[0] + 1] = Point(center_x                 , center_y + 2*radius);
	pts[npts[0] + 2] = Point(center_x - radius*1.73205, center_y - radius  );

	// copying supertriangle points to device
	hipMemcpy(&(pts_d[npts[0]]), &(pts[npts[0]]), 3 * sizeof(Point), hipMemcpyHostToDevice);
	hipFree(avgPoint_d);
	hipFree(largest_dist_d);

	// writing supertriangle on host
	int p[3] = {npts[0], npts[0]+1, npts[0]+2};
	int n[3] = {-1, -1, -1}; 
	int o[3] = {-1, -1, -1}; 

	// writing supertriangle on host
	writeTri(pts, &(triList[0]), p, n, o);

	memset(ptToTri, 0, npts[0] * sizeof(int));
	hipMemset(ptToTri_d, 0, npts[0] * sizeof(int));

	hipMemcpy(&(triList_d[0]), &(triList[0]), sizeof(Tri), hipMemcpyHostToDevice);
	//setPtsAll<<<1, 1>>>(npts_d, triList_d);

	(*nTri)++;
	hipMemcpy(nTri_d, nTri, sizeof(int), hipMemcpyHostToDevice);
}

void Delaunay::cpyToHost() {
	hipMemcpy(triList_d, triList, nTri[0] * sizeof(Tri), hipMemcpyDeviceToHost);
}

/* ============================================= GPU CODE ============================================= */

//void Delaunay::gpu_compute() {}


/*
 */
__global__ void sumPoints(Point* pts_d, int npts, Point *avgPoint_d) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < 2*npts) {
		atomicAddFloat(&(avgPoint_d->x[idx%2]), pts_d[idx/2].x[idx%2]);
	}
}

/*
 * Sets the spts (search points) of the chosen triangle to be all of the points 
 * of the final triangulation.
 */
//__global__ void setSptsAll(int npts, Tri* triList_d, int i) {
//	int idx = blockIdx.x * blockDim.x + threadIdx.x;
//
//	if (idx == 0) {
//		triList_d[i].nspts = npts;
//
//		triList_d[i].spts = new int[npts];
//		for (int i=0; i<npts; ++i) {
//			triList_d[i].spts[i] = i;
//		}
//	}
//}
__global__ void computeMaxDistPts(Point* pts_d, int npts, float* largest_dist_d) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int i, j;
	float dist; 

	int count = (npts*(npts - 1)) / 2; 
	if (idx < count) {
		// for now yoinnked from chet gipiti but uses triangilar number math to get convenient indexing
		i = (int)((2*npts - 1 - sqrtf((2*npts - 1) * (2*npts - 1) - 8*idx)) / 2);
		j = idx - (i*(2*npts - i - 1) / 2) + i + 1;

		printf("index: %d | count: %d | i,j =(%d, %d)\n", idx, count, i, j);

		dist = sqrtf( (pts_d[i].x[0] - pts_d[j].x[0])*(pts_d[i].x[0] - pts_d[j].x[0]) +
				      (pts_d[i].x[1] - pts_d[j].x[1])*(pts_d[i].x[1] - pts_d[j].x[1]));

		if (dist > (*largest_dist_d)) {
			atomicMaxFloat(largest_dist_d, dist); 
		}
	}
}

//__global__ void writeTriKernel(Tri* tri, int* p, int* n, int* o) {
//	writeTri(tri, p, n, o);
//}

__host__ __device__ void writeTri(Point* pts, Tri* tri, int* p, int* n, int* o) {
	for (int i=0; i<3; i++) {
		tri->p[i] = p[i];
		tri->n[i] = n[i];
		tri->o[i] = o[i];
	}

	tri->flip = -1;
	tri->insertPt = -1;
	//tri->insertPt_dist = ;

}

//__global__ void writeUnisertedPts(Point* pts_d, int npts, Tri* triList) {

/*
 * Writes the shortest distance bewteen a point in each triangle and the center
 * of its circumcircle
 */
__global__ void setInsertPtsDistance(Point* pts, int npts, Tri* triList) {
	int idx = blockidx.x * blockdim.x + threadidx.x;

	if (idx < npts) {
		if (ptToTri[idx] >= 0) {      // for uninserted points
			int idxTri = ptToTri[idx];

			Point center;
			circumcircle_center(pts[triList[idxTri].p[0]], 
					            pts[triList[idxTri].p[1]],
					            pts[triList[idxTri].p[2]],
					            &center);

			triList[idxTri].circumcenter.x[0] = center.x[0]; 
			triList[idxTri].circumcenter.x[1] = center.x[1]; 

			float dist = dist(center, pts[idx]); 
			atomicMinFloat(&(triList[idxTri].insertPt_dist), dist);
	}
}
	
__global__ void setInsertPts(Point* pts, int npts, Tri* triList) {
	int idx = blockidx.x * blockdim.x + threadidx.x;

	if (idx < npts) {
		if (ptToTri[idx] >= 0) {      // for uninserted points
			int idxTri = ptToTri[idx];

			float dist = dist(pts[triList[idxTri].insertPt[0]], pts[idx]); 
			if (dist == triList[idxTri].insertPt_dist) {
				atomicExch(&(triList[idxTri].insertPt), idx);
			}
		}
	}
}

void Delaunay::prepForInsert() {

	dim3 threadsPerBlock(32);
	dim3 numBlocks((*npts)/threadsPerBlock.x + (!((*npts) % threadsPerBlock.x) ? 0:1));

	setInsertPtsDistance<<<numBlocks, threadsPerBlock>>>(pts_d, *npts, triList_d);
	//setInsertPts<<<numBlocks, threadsPerBlock>>>(pts_d, *npts, triList_d);
}


__global__ void insert(Tri* triList_d, int* nTri_d) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < npts) {
		// find which tri point lies in 
		int idxTri = ptToTri[idx];
		triList[idxTri].center = ;
	}
}


//__global__ checkInsert_gpu() {
//	int idx = blockIdx.x * blockDim.x + threadIdx.x;
//
//	num_tris_to_insert_d = 0;
//	if (idx < nTri_d) {
//		if (triList_d[idx].spts_alloc == true) { // triList[i].nspts > 0 && 
//			triList_d[idx].get_center();
//			atomicAdd(&num_tris_to_insert_d, 1);
//		}
//	}
//}
//
//__global__ insert_gpu() {
//	int idx = blockIdx.x * blockDim.x + threadIdx.x;
//
//	//int num_inserted_tri = 0;
//	//int max = nTri;
//	if (idx<max) {
//		insertInTri(idx);
//	}
//}


/* ====================================================================================================== */

__device__ float atomicAddFloat(float* address, float val) {
	int* address_as_ull = (int*)address;
	int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __float_as_int(val + __int_as_float(assumed)));
	} while (assumed != old);
	return __int_as_float(old);
}

__device__ float atomicMaxFloat(float* address, float val) {
	int* address_as_ull = (int*)address;
	int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __float_as_int(max(val, __int_as_float(assumed))));
	} while (assumed != old);
	return __int_as_float(old);
}

__device__ float atomicMinFloat(float* address, float val) {
	int* address_as_ull = (int*)address;
	int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __float_as_int(min(val, __int_as_float(assumed))));
	} while (assumed != old);
	return __int_as_float(old);
}

void Delaunay::saveToFile() {

	saveFile << iter << " " << nTri[0] << "\n";
	for (int i=0; i<nTri[0]; ++i) {
		for (int j=0; j<3; ++j) {
			saveFile << triList[i].p[j] << " "; 
		} 
		for (int j=0; j<3; ++j) {
			saveFile << triList[i].n[j] << " "; 
		} 
		for (int j=0; j<3; ++j) {
			saveFile << triList[i].o[j] << " "; 
		} 
		saveFile << triList[i].flip << " "; 

		saveFile << "\n"; 
	}

	saveFile << "\n"; 
	iter++;
}
