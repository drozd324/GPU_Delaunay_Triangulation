#include "hip/hip_runtime.h"
#include "atomic.h"

__device__ float atomicAddFloat(float* address, float val) {
	int* address_as_ull = (int*)address;
	int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __float_as_int(val + __int_as_float(assumed)));
	} while (assumed != old);
	return __int_as_float(old);
}

__device__ float atomicMaxFloat(float* address, float val) {
	int* address_as_ull = (int*)address;
	int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __float_as_int(max(val, __int_as_float(assumed))));
	} while (assumed != old);
	return __int_as_float(old);
}

__device__ float atomicMinFloat(float* address, float val) {
	int* address_as_ull = (int*)address;
	int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __float_as_int(min(val, __int_as_float(assumed))));
	} while (assumed != old);
	return __int_as_float(old);
}




__device__ double atomicAddDouble(double* address, double val) {
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
	} while (assumed != old);
	return __longlong_as_double(old);
}

__device__ double atomicMaxDouble(double* address, double val) {
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,__double_as_longlong(max(val,__longlong_as_double(assumed))));
	} while (assumed != old);
	return __longlong_as_double(old);
}

__device__ double atomicMinDouble(double* address, double val) {
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,__double_as_longlong(min(val,__longlong_as_double(assumed))));
	} while (assumed != old);
	return __longlong_as_double(old);
}

