
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 10000

__global__ void my_atomic(int* vect, int* sum) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) {
        atomicAdd(sum, vect[idx]);
    }
}

int main() {
    int sum[1] = {0};
    int* a = (int*)malloc(N * sizeof(int));

    int expected_sum = 0;
    for (int i = 0; i < N; i++) {
        a[i] = i;
        expected_sum += i;
    }

    int *a_d, *sum_d;
    hipMalloc(&a_d, N * sizeof(int));
    hipMalloc(&sum_d, sizeof(int));

    hipMemcpy(a_d, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(sum_d, sum, sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x);

    printf("Before: %d\n", sum[0]);

    my_atomic<<<numBlocks, threadsPerBlock>>>(a_d, sum_d);
    hipDeviceSynchronize();

    hipMemcpy(sum, sum_d, sizeof(int), hipMemcpyDeviceToHost);
    printf("After: %d (Expected: %d)\n", sum[0], expected_sum);

    hipFree(a_d);
    hipFree(sum_d);
    free(a);

    return 0;
}

