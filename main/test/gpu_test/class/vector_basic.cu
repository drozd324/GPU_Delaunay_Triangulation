
#include <hip/hip_runtime.h>
#include <stdio.h>

// Size of array
#define N 10

// Kernel
__global__ void add_vectors(double* a, double* b, double* c) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id < N) { 
		c[id] = a[id] + b[id];
	}
}

void print_array(double* array, int n) {
	for (int i=0; i<n; ++i) {
		printf("%lf ", array[i]);
	}
	printf("\n");
}

// Main program
int main() {
    // Number of bytes to allocate for N doubles

    double *a = (double*)malloc(N * sizeof(double));
    double *b = (double*)malloc(N * sizeof(double));
    double *c = (double*)malloc(N * sizeof(double));

    for(int i=0; i<N; i++) {
        a[i] = 1.0;
        b[i] = 2.0;
    }

    double *d_a, *d_b, *d_c;
    hipMalloc(&d_a, N * sizeof(double));
    hipMalloc(&d_b, N * sizeof(double));
    hipMalloc(&d_c, N * sizeof(double));

    hipMemcpy(d_a, a, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(double), hipMemcpyHostToDevice);

	dim3 threadsPerBlock(32);
    dim3 numBlocks(N/threadsPerBlock.x + (!(N % threadsPerBlock.x) ? 0:1));
    add_vectors<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, N * sizeof(double), hipMemcpyDeviceToHost);

	print_array(a, N);
	print_array(b, N);
	print_array(c, N);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(a);
    free(b);
    free(c);

    printf("\n---------------------------\n");
    printf("__SUCCESS__\n");
    printf("---------------------------\n");
    printf("N                 = %d\n", N);
    printf("Threads Per Block = %d\n", threadsPerBlock);
    printf("Blocks In Grid    = %d\n", numBlocks);
    printf("---------------------------\n\n");

    return 0;
}
